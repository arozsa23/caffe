
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_conv_layer.hpp"

namespace caffe {

__global__ void sync_conv_groups() { }

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;

  // *****************************************************************************
  // BANG
  const Dtype beta = this->layer_param_.convolution_param().beta();
  const Dtype epsilon = this->layer_param_.convolution_param().epsilon();
  const Dtype ratio = this->layer_param_.convolution_param().ratio();
  const vector<bool>& classifications = Caffe::classifications();
  // *****************************************************************************

  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Weight diffs (including bias): Based on scaled original top_diff if necessary

    // *****************************************************************************
    // BANG: re-scaling top_diff based on L-2 norms of batch elements
    Dtype* mutable_top_diff = top[i]->mutable_gpu_diff();

    if (beta != 0.0) {
      int batch_size = top[i]->shape(0);
      // Storing L-2 norms for elements of the batch
      vector<Dtype> l2_norms(batch_size);
      Dtype max_l2 = 0.0;

      for (int j = 0; j < batch_size; ++j) {
        Dtype diff_l2;
        caffe_gpu_dot(this->top_dim_, top_diff + j * this->top_dim_,
                      top_diff + j * this->top_dim_, &diff_l2);
        diff_l2 = std::sqrt(diff_l2);
        max_l2 = std::max(max_l2, diff_l2);
        l2_norms[j] = diff_l2;
      }

      // Scaling
      for (int j = 0; j < batch_size; ++j) {
        if (l2_norms[j] != 0.0) {
          Dtype eps = epsilon;
          if (classifications[j] == 0) {
            // Incorrectly classified sample
            eps = epsilon * ratio;
          }
          const Dtype scale = pow( max_l2 / l2_norms[j], eps * (1. - l2_norms[j] / max_l2));
          caffe_gpu_scal(this->top_dim_, scale, mutable_top_diff + j * this->top_dim_);
        }
      }
    }
    // *****************************************************************************

    for (int g = 0; g < this->group_; g++) {

      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
      }

    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }

  // *****************************************************************************
  // BANG: applying local learning rate (beta of BANG)
  if (beta != 0.0) {
    // Scale bias_diff with beta
    this->blobs_[1]->scale_diff(beta);
    // Scale weight_diff with beta
    this->blobs_[0]->scale_diff(beta);
  }
  // *****************************************************************************
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
